#include "hip/hip_runtime.h"
/* When compiling the CUDA code, we do not want to include all ecm-impl.h*/
#define _DO_NOT_INCLUDE_ECM_IMPL_H
#include "ecm-gpu.h"
#include <gmp.h>
#include "cudakernel.h"

#ifndef __HIPCC__
#error "This file should only be compiled with nvcc"
#endif

__constant__ __device__ digit_t d_invNcst;
__device__ biguint_t d_Ncst;
__device__ biguint_t d_3Ncst;
__device__ biguint_t d_Mcst;


#define errCheck(err) cuda_errCheck (err, __FILE__, __LINE__)
#define hipMalloc(d, size) errCheck (hipMalloc (d, size))
#define cudaMemcpyHtoD(d, h, size) errCheck (hipMemcpy ((void *) d, \
                                    (void *) h, size, hipMemcpyHostToDevice))
#define cudaMemcpyDtoH(h, d, size) errCheck (hipMemcpy ((void *) h, \
                                    (void *) d, size, hipMemcpyDeviceToHost))
#define cudaMemcpyCst(d, h, size) errCheck (hipMemcpyToSymbol(HIP_SYMBOL(d), h, size))


/******************************/
/* Host code handling the GPU */
/******************************/

inline void cuda_errCheck (hipError_t err, const char *file, const int line)
{
  if( err != hipSuccess ) 
  {
    fprintf(stderr, "%s(%i) : Error cuda : %s.\n",
              file, line, hipGetErrorString( err) );
    exit(EXIT_FAILURE);
  }
}

/* First call to a global function initialize the device */
__global__ void Cuda_Init_Device ()
{
}

/* Given the compute compatibility (as major.minor), return the number of block
 * to be run on one multiprocessor. */
extern "C"
unsigned int
getNumberOfBlockPerMultiProcessor (int major, int minor)
{
  /* For 2.0 and 2.1, limited by the maximum number of threads per MP and the
   * number of available registrer (need 23 registers per threads).
   */
  if (major == 2)
    return 1;
  /* For 3.0, 3.2, 3.5 and 3.7 limited by the maximum number of threads per MP.
   */
  else if (major == 3)
    return 2;
  /* For 5.0, 5.2, and 5.3 limited by the maximum number of threads per MP. */
  else if (major == 5)
    return 2;
  /* We assume that for newer compute capability the properties of the GPU won't
   * decrease.
   */
  else
    return 2;
}

extern "C" 
int 
select_and_init_GPU (int device, unsigned int *number_of_curves, int verbose)
{
  hipDeviceProp_t deviceProp;
  hipError_t err;
        
  if (device!=-1)
    {
      if (verbose)
          fprintf (stdout, "GPU: device %d is required.\n", device);

      err = hipSetDevice(device);
      if (err != hipSuccess)
        {
          fprintf (stderr, "GPU: Error: Could not use device %d\n", device);
          fprintf (stderr, "GPU: Error msg: %s\n", hipGetErrorString(err));
          return -1;
        }
    }
  
  err = hipGetDevice (&device);
  if (err != hipSuccess)
    {
      fprintf (stderr, "GPU: Error: no active device.\n");
      fprintf (stderr, "GPU: Error msg: %s\n", hipGetErrorString(err));
      return -1;
    }

  err = hipGetDeviceProperties (&deviceProp, device);
  if (err != hipSuccess)
    {
      fprintf (stderr, "GPU: Error while getting device's properties.\n");
      fprintf (stderr, "GPU: Error msg: %s\n", hipGetErrorString(err));
      return -1;
    }

  if (verbose)
    {
      printf ("GPU: will use device %d: %s, compute capability %d.%d, %d MPs.\n"
              "GPU: maxSharedPerBlock = %zu maxThreadsPerBlock = %d "
              "maxRegsPerBlock = %d\n", device, deviceProp.name,
              deviceProp.major, deviceProp.minor,
              deviceProp.multiProcessorCount, deviceProp.sharedMemPerBlock,
              deviceProp.maxThreadsPerBlock, deviceProp.regsPerBlock);
    }


  if (*number_of_curves == 0) /* if choose the number of curves */
    {
      unsigned int n, m = ECM_GPU_CURVES_BY_BLOCK;
      n = getNumberOfBlockPerMultiProcessor (deviceProp.major, deviceProp.minor);
      *number_of_curves = n * deviceProp.multiProcessorCount * m;
    }
  else if (*number_of_curves % ECM_GPU_CURVES_BY_BLOCK != 0)
    {
      /* number_of_curves should be a multiple of ECM_GPU_CURVES_BY_BLOCK */
      *number_of_curves = (*number_of_curves / ECM_GPU_CURVES_BY_BLOCK + 1) * 
                                                        ECM_GPU_CURVES_BY_BLOCK;
      if (verbose)
          fprintf(stderr, "GPU: the requested number of curves has been "
                          "modified to %u\n", *number_of_curves);
    }

  /* First call to a global function initialize the device */
  errCheck (hipSetDeviceFlags (hipDeviceScheduleYield)); 
  Cuda_Init_Device<<<1, 1>>> ();
  errCheck (hipGetLastError()); 

  if (verbose)
  {
    struct hipFuncAttributes kernelAttr;
    err = hipFuncGetAttributes(&kernelAttr, reinterpret_cast<const void*>(Cuda_Ell_DblAdd));
    if (err == hipSuccess)
    {
      printf ("GPU: Using device code targeted for architecture compile_%d\n"
              "GPU: Ptx version is %d\nGPU: maxThreadsPerBlock = %d\n"
              "GPU: numRegsPerThread = %d sharedMemPerBlock = %zu bytes\n",
              kernelAttr.binaryVersion, kernelAttr.ptxVersion,
              kernelAttr.maxThreadsPerBlock, kernelAttr.numRegs,
              kernelAttr.sharedSizeBytes);
    }
  }

  return 0;
}

extern "C"
float cuda_Main (biguint_t h_N, biguint_t h_3N, biguint_t h_M, digit_t h_invN,
                 biguint_t *h_xarray, biguint_t *h_zarray,
                 biguint_t *h_x2array, biguint_t *h_z2array, mpz_t s,
                 unsigned int firstinvd, unsigned int number_of_curves,
                 int verbose)
{
  hipEvent_t start, stop;
  hipEventCreate (&start);
  hipEventCreate (&stop);
  hipEventRecord (start, 0);

  size_t j;
  int i;
  float elltime = 0.0;
  biguint_t *d_xA, *d_zA, *d_xB, *d_zB;

#define MAXEVENTS 2
#define DEPTH_EVENT 32
  hipEvent_t event[MAXEVENTS];   // Space for some cuda Event Handles
  long nEventsRecorded = 0;   // Remember how many events are recorded
  long eventrecordix = 0;     // Remember index of next event to record
  long eventsyncix;       // Remember index of oldest recorded event

  size_t array_size = sizeof(biguint_t) * number_of_curves;

  dim3 dimBlock (ECM_GPU_NB_DIGITS, ECM_GPU_CURVES_BY_BLOCK);
  dim3 dimGrid (number_of_curves/ ECM_GPU_CURVES_BY_BLOCK);
  if (verbose)
    {
      fprintf(stdout, "GPU: Block: %ux%ux%u Grid: %ux%ux%u "
              "(%d parallel curves)\n", dimBlock.x, dimBlock.y, dimBlock.z,
              dimGrid.x, dimGrid.y, dimGrid.z, number_of_curves);
    }

  /* Create a pair of events to pace ourselves */
  for (i=0; i<MAXEVENTS; i++)
    errCheck (hipEventCreateWithFlags (&event[i], 
                              hipEventBlockingSync|hipEventDisableTiming));

  hipMalloc (&d_xA, array_size);
  hipMalloc (&d_zA, array_size);
  hipMalloc (&d_xB, array_size);
  hipMalloc (&d_zB, array_size);

  /* Copy into the gpu memory */
  cudaMemcpyCst (d_invNcst, (void *) &h_invN, sizeof(digit_t));
  cudaMemcpyCst (d_Ncst, (void *) h_N, sizeof(biguint_t));
  cudaMemcpyCst (d_3Ncst, (void *) h_3N, sizeof(biguint_t));
  cudaMemcpyCst (d_Mcst, (void *) h_M, sizeof(biguint_t));

  cudaMemcpyHtoD (d_xA, h_xarray, array_size);
  cudaMemcpyHtoD (d_zA, h_zarray, array_size);
  cudaMemcpyHtoD (d_xB, h_x2array, array_size);
  cudaMemcpyHtoD (d_zB, h_z2array, array_size);

#ifdef PRINT_REMAINING_ITER
      unsigned int jmod = 100000000;
#endif

  /* Double-and-add loop: it calls the GPU for each bits of s */
  for (j = mpz_sizeinbase (s, 2) - 1; j>0; j-- )
  {
    if (mpz_tstbit (s, j-1) == 1)
      Cuda_Ell_DblAdd<<<dimGrid,dimBlock>>>(d_xB, d_zB, d_xA, d_zA, firstinvd);
    else
      Cuda_Ell_DblAdd<<<dimGrid,dimBlock>>>(d_xA, d_zA, d_xB, d_zB, firstinvd);

    /* Pace entry of events. Less overhead to enter an event every few    */
    /* iterations. But, if you exceed the depth of NVIDIA's kernel queue, */
    /* it will busy-loop!                                                 */
    /* Enter an event every DEPTH_EVENT iteration */
    if (j % DEPTH_EVENT == 0)  
    {
      hipEventRecord(event[eventrecordix]); 
      if (nEventsRecorded == 0)     
        eventsyncix = eventrecordix; 
      nEventsRecorded += 1;          
      eventrecordix = (eventrecordix+1)%MAXEVENTS;  
    }

    if (nEventsRecorded == MAXEVENTS) 
    {
      hipEventSynchronize(event[eventsyncix]);  
      nEventsRecorded -= 1;   
      eventsyncix = (eventsyncix+1)%MAXEVENTS; 
    }

#ifdef PRINT_REMAINING_ITER
    if (j < 100000000) jmod = 10000000;
    if (j < 10000000)  jmod =  1000000;
    if (j < 1000000)   jmod =   100000;
    if (j < 100000)    jmod =    10000;
    if (j % jmod == 0)
      printf("%lu iterations to go\n", j);
#endif
  }

  /* If an error occurs during the kernel calls in the loop */
  errCheck (hipGetLastError()); 

  /* Await for last recorded events */
  while (nEventsRecorded != 0) 
  {
    hipEventSynchronize(event[eventsyncix]); 
    nEventsRecorded -= 1;          
    eventsyncix = (eventsyncix+1)%MAXEVENTS; 
  }

  /* Get the results back from device memory */
  cudaMemcpyDtoH (h_xarray, d_xA, array_size);
  cudaMemcpyDtoH (h_zarray, d_zA, array_size);

  /* Clean up our events and our stream handle */
  for (i=0; i<MAXEVENTS; i++)
    errCheck (hipEventDestroy(event[i]));


  hipFree ((void *) d_xA);
  hipFree ((void *) d_zA);
  hipFree ((void *) d_xB);
  hipFree ((void *) d_zB);

  hipEventRecord (stop, 0);
  hipEventSynchronize (stop);

  hipEventElapsedTime (&elltime, start, stop);

  errCheck (hipEventDestroy (start));
  errCheck (hipEventDestroy (stop));

  return elltime;
}



/***************/
/* Device code */
/***************/

#if defined(_MSC_VER)
#  define ASM asm volatile
#else
#  define ASM asm __volatile__
#endif

#define __add_cc(r,a,b) ASM ("add.cc.u32 %0, %1, %2;": "=r"(r): "r"(a), "r"(b)) 
#define __addc_cc(r,a,b) ASM ("addc.cc.u32 %0, %1, %2;": "=r"(r): "r"(a), "r"(b))
#define __sub_cc(r,a,b) ASM ("sub.cc.u32 %0, %1, %2;": "=r"(r): "r"(a), "r"(b)) 

#define __addcy(carry) ASM ("addc.s32 %0, 0, 0;": "=r"(carry)) 
#define __addcy2(carry) ASM ("addc.cc.s32 %0, %0, 0;": "+r"(carry)) 

#define __subcy(carry) ASM ("subc.s32 %0, 0, 0;": "=r"(carry)) 
#define __subcy2(carry) ASM ("subc.s32 %0, %0, 0;": "+r"(carry)) 

#define __mul_lo(r,a,b) ASM("mul.lo.u32 %0, %1, %2;": "=r"(r): "r"(a),"r"(b)) 
#define __mul_hi(r,a,b) ASM("mul.hi.u32 %0, %1, %2;": "=r"(r): "r"(a),"r"(b)) 
#define __mad_lo_cc(r,a,b) ASM("mad.lo.cc.u32 %0, %1, %2, %0;":\
                                                      "+r"(r): "r"(a),"r"(b)) 
#define __madc_hi_cc(r,a,b) ASM("madc.hi.cc.u32 %0, %1, %2, %0;":\
                                                  "+r"(r):"r"(a),"r"(b)) 

#ifdef __CUDA_ARCH__
  #if __CUDA_ARCH__ >= 200
    #include "cudakernel_default.cu"
  #else
    #error "Unsupported architecture"
  #endif
#endif
