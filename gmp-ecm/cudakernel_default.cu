#include "hip/hip_runtime.h"
/*            Default code for GPU                  */
/* A compute capability of 2.0 at least is required */


__device__ void Cuda_Fully_Normalize (biguint_t A, bigint_t cy)
{
  carry_t cytemp;
  unsigned int thm1;

  while(__any(cy[threadIdx.x])!=0)
  {
    thm1 = (threadIdx.x - 1) % ECM_GPU_NB_DIGITS;
    cytemp = cy[thm1];

    __add_cc(A[threadIdx.x], A[threadIdx.x], cytemp);
  
    if (cytemp >= 0)
      __addcy(cy[threadIdx.x]);
    else /* if (cytemp < 0) */
      __subcy(cy[threadIdx.x]);
  }
}

/* Compute Rmod <- A + B */ 
/* Input: 0 <= A, B < 3*N */ 
/* Ouput: 0 <= Rmod < 6*N */ 
__device__ void Cuda_Add_mod
(biguint_t Rmod, bigint_t cy, const biguint_t A, const biguint_t B)
{
  unsigned int thp1 = (threadIdx.x + 1) % ECM_GPU_NB_DIGITS;
  __add_cc (Rmod[threadIdx.x], A[threadIdx.x], B[threadIdx.x]);
  __addcy2(Rmod[thp1]); 
  __addcy (cy[thp1]);
  Cuda_Fully_Normalize (Rmod, cy); 
}

/* Compute Rmod <- Rmod + B */ 
/* Input: 0 <= Rmod, B < 3*N */ 
/* (except when it follows Cuda_Mulint_mod, 0 <= Rmod < 3*N, 0 < B < 7*N ) */ 
/* Ouput: 0 <= Rmod < 6*N */ 
/* (except when it follows Cuda_Mulint_mod, 0 <= Rmod < 10*N) */ 
__device__ void Cuda_Add_mod
(biguint_t Rmod, bigint_t cy, const biguint_t A)
{
  unsigned int thp1 = (threadIdx.x + 1) % ECM_GPU_NB_DIGITS;
  __add_cc (Rmod[threadIdx.x], Rmod[threadIdx.x], A[threadIdx.x]);
  //__addcy (cy[threadIdx.x]);
  __addcy2(Rmod[thp1]); 
  __addcy (cy[thp1]);
  Cuda_Fully_Normalize (Rmod, cy);
}

/* Compute Rmod <- Rmod - B */ 
/* Input: 0 <= Rmod, B < 3*N */ 
/* Ouput: 0 <= Rmod < 6*N */ 
__device__ void Cuda_Sub_mod 
(biguint_t Rmod, bigint_t cy, const biguint_t B, const digit_t N3thdx)
{
  digit_t reg_Rmod = Rmod[threadIdx.x];
  carry_t reg_cy = 0; 
  
  __add_cc (reg_Rmod, reg_Rmod, N3thdx);
  __addcy (reg_cy);
  __sub_cc (reg_Rmod, reg_Rmod, B[threadIdx.x]);
  __subcy2 (reg_cy);

  Rmod[threadIdx.x] = reg_Rmod;
  cy[threadIdx.x] = reg_cy;
  Cuda_Fully_Normalize (Rmod, cy); 
}

/* Perform one step of REDC */ 
__device__ void Cuda_Mulmod_step
(biguint_t r, bigint_t cy, digit_t a, digit_t b, const digit_t Nthdx,
 const digit_t invN)
{
  digit_t t;
  digit_t reg_hi = 0;
  unsigned int thp1= (threadIdx.x + 1) % ECM_GPU_NB_DIGITS;
  carry_t reg_cy = cy[thp1];

  __mad_lo_cc(r[threadIdx.x],a,b);
  __madc_hi_cc(reg_hi,a,b);
  __addcy2(reg_cy);

  __mul_lo(t, invN, r[0]);
  __mad_lo_cc(r[threadIdx.x],t,Nthdx);
  __madc_hi_cc(reg_hi,t,Nthdx);
  __addcy2(reg_cy);

  /* make one round of normalize + a right shift at the same time */
  __add_cc(r[threadIdx.x],r[thp1],reg_hi);
  __addc_cc(r[thp1],r[thp1],reg_cy);
  __addcy(cy[thp1]); 
}

/* Compute r <- 2*a */ 
/* Input: 0 <= a < 3*N */ 
/* Ouput: 0 <= r < 3*N */ 
__device__ void Cuda_Dbl_mod
(biguint_t r, biguint_t a)
{
  unsigned int thp1= (threadIdx.x + 1) % ECM_GPU_NB_DIGITS;
  asm ("add.cc.u32 %0, %1, %1;" : "=r"(r[threadIdx.x]) : "r"(a[threadIdx.x]));
  __addcy2(r[thp1]);
}


/* Compute r <- A*b */ 
/* Input: 0 < b < 2^SIZE_DIGIT, 0 <= A < 6*N */ 
/* Ouput: 0 <= r < 7*N */ 
__device__ void Cuda_Mulint_mod
(biguint_t r, bigint_t cy, biguint_t A, digit_t b, const digit_t Nthdx,
 const digit_t invN)
{
  digit_t t;
  digit_t reg_hi;
  unsigned int thp1= (threadIdx.x + 1) % ECM_GPU_NB_DIGITS;
  digit_t reg_A = A[threadIdx.x];
  carry_t reg_cy;

  __mul_lo(r[threadIdx.x],reg_A,b);
  __mul_hi(reg_hi,reg_A,b);

  __mul_lo(t, invN, r[0]);
  __mad_lo_cc(r[threadIdx.x],t,Nthdx);
  __madc_hi_cc(reg_hi,t,Nthdx);
  __addcy(reg_cy);

  /* make one round of normalize + a right shift at the same time */
  __add_cc(r[threadIdx.x],r[thp1],reg_hi);
  __addc_cc(r[thp1],r[thp1],reg_cy);
  __addcy(cy[thp1]); 

  Cuda_Fully_Normalize(r,cy); 
}

/* Compute r <- A*B */ 
/* Input: 0 <= A, B < 6*N */
/* (except when it follows Cuda_Mulint_mod, 0 <= A < 6*N, 0 < B < 10*N ) */ 
/* Ouput: 0 <= r < 3*N */ 
__device__ void Cuda_Mul_mod 
(biguint_t mul, bigint_t cy, const biguint_t A, const biguint_t B, biguint_t r,
 const digit_t Nthdx, const digit_t invN)
{

  int i;
  digit_t temp=A[threadIdx.x];

  r[threadIdx.x]=0;
  
  for (i=0; i<ECM_GPU_NB_DIGITS; i++)
    Cuda_Mulmod_step (r, cy, temp, B[i], Nthdx, invN);

  
  Cuda_Fully_Normalize (r, cy);
  mul[threadIdx.x]=r[threadIdx.x];
}

__device__ void Cuda_Square_mod 
(biguint_t mul, bigint_t cy, const biguint_t A, biguint_t r, 
 const digit_t Nthdx, const digit_t invN)
{
  Cuda_Mul_mod (mul, cy, A, A, r, Nthdx, invN);
}

/* 
  Compute silmutaneously:
  (xarg : zarg ) <- [2](xarg : zarg) 
  (xarg2 : zarg2 ) <- (xarg : zarg) + (xarg2 : zarg2) 
*/
__global__ void 
Cuda_Ell_DblAdd (biguint_t *xAarg, biguint_t *zAarg, biguint_t *xBarg, 
                                       biguint_t *zBarg, unsigned int firstinvd)
{
  __shared__ VOL digit_t b_temp_r[ECM_GPU_CURVES_BY_BLOCK][ECM_GPU_NB_DIGITS];
  __shared__ VOL carry_t b_cy[ECM_GPU_CURVES_BY_BLOCK][ECM_GPU_NB_DIGITS]; 

  __shared__ VOL digit_t b_t[ECM_GPU_CURVES_BY_BLOCK][ECM_GPU_NB_DIGITS];
  __shared__ VOL digit_t b_u[ECM_GPU_CURVES_BY_BLOCK][ECM_GPU_NB_DIGITS];
  __shared__ VOL digit_t b_v[ECM_GPU_CURVES_BY_BLOCK][ECM_GPU_NB_DIGITS];
  __shared__ VOL digit_t b_w[ECM_GPU_CURVES_BY_BLOCK][ECM_GPU_NB_DIGITS];
  
  VOL digit_t *t=b_t[threadIdx.y];
  VOL digit_t *u=b_u[threadIdx.y];
  VOL digit_t *v=b_v[threadIdx.y];
  VOL digit_t *w=b_w[threadIdx.y];
  VOL digit_t *temp_r=b_temp_r[threadIdx.y];
  VOL carry_t *cy=b_cy[threadIdx.y];

  /* Init of shared variables */
  const unsigned int idx1=blockIdx.x*blockDim.y+threadIdx.y;
  //unsigned int t1=threadIdx.x+1;
  cy[threadIdx.x]=0; 

  w[threadIdx.x]=xBarg[idx1][threadIdx.x];
  v[threadIdx.x]=zBarg[idx1][threadIdx.x];
  temp_r[threadIdx.x]=xAarg[idx1][threadIdx.x];
  u[threadIdx.x]=zAarg[idx1][threadIdx.x];

  const digit_t Nthdx = d_Ncst[threadIdx.x]; 
  const digit_t N3thdx = d_3Ncst[threadIdx.x]; 
  const digit_t invN = d_invNcst; 

  Cuda_Add_mod(t, cy, v, w);           /* C=zB+xB */
  Cuda_Sub_mod(v, cy, w, N3thdx);      /* D=zB-xB */
  Cuda_Add_mod(w, cy, u, temp_r);      /* A=zA+xA */
  Cuda_Sub_mod(u, cy, temp_r, N3thdx); /* B=zA-xA */

  Cuda_Mul_mod(t, cy, t, u, temp_r, Nthdx, invN); /* CB=C*B=(zB+xB)(zA-xA) */
  Cuda_Mul_mod(v, cy, v, w, temp_r, Nthdx, invN); /* DA=D*A=(zB-xB)(zA+xA) */

  Cuda_Square_mod(w, cy, w, temp_r, Nthdx, invN); /* AA=A^2 */
  Cuda_Square_mod(u, cy, u, temp_r, Nthdx, invN); /* BB=B^2 */

  Cuda_Mul_mod(temp_r, cy, u, w, temp_r, Nthdx, invN); /* AA*BB */
  xAarg[idx1][threadIdx.x]=temp_r[threadIdx.x];

  Cuda_Sub_mod (w, cy, u, N3thdx); /* K= AA-BB */
  Cuda_Mulint_mod (temp_r, cy, w, idx1 + firstinvd, Nthdx, invN); /* d*K */ 
  Cuda_Add_mod (u, cy, temp_r); /* BB+d*K */
 
  Cuda_Mul_mod (w, cy, w, u, temp_r, Nthdx, invN); /* K*(BB+d*K) */
  zAarg[idx1][threadIdx.x]=w[threadIdx.x];
 
  Cuda_Add_mod(w, cy, v, t);       /* DA+CB mod N */
  Cuda_Sub_mod(v, cy, t, N3thdx);  /* DA-CB mod N */

  Cuda_Square_mod(w, cy, w, temp_r, Nthdx, invN); /* (DA+CB)^2 mod N */
  Cuda_Square_mod(v, cy, v, temp_r, Nthdx, invN); /* (DA-CB)^2 mod N */

  /* z0=1 so there is nothing to compute for z0*(DA+CB)^2 */
  Cuda_Dbl_mod(temp_r, v); /* x0=2 x0*(DA-CB)^2 */
  
  xBarg[idx1][threadIdx.x]=w[threadIdx.x];
  zBarg[idx1][threadIdx.x]=temp_r[threadIdx.x];
}

